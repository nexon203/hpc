#include "hip/hip_runtime.h"
/****************************************************************************
  Similar to factorise_3_0 but solves the problem with 4 threads using a 
  block method for search space partitioning. It is included here to 
  accompany a CUDA version of the program.
./pswcuda > result.txt

  Compile with:

   nvcc -o pswcuda pswcuda.cu
  
  Dr Kevan Buckley, University of Wolverhampton, 2018
*****************************************************************************/

#include <stdio.h>
#include <hip/hip_runtime_api.h>
#include <time.h>

__device__ int is_a_match(char *password){

	char password1[]="NT6745";
	char password2[]="BJ1945";
	char password3[]="PD5612";
	char password4[]="RT7856";

	
	char *p1 = password;
	char *p2 = password;
	char *p3 = password;
	char *p4 = password;

	
	char *w1 = password1;
	char *w2 = password2;
	char *w3 = password3;
	char *w4 = password4;


	while(*p1 == *w1){


	if(*p1 == '\0'){

	printf("Password found : %s\n", password1);
	return 1;

}
p1++;
w1++;



}


while(*p2 == *w2){


	if(*p2 == '\0'){

	printf("Password found : %s\n", password2);
	return 1;

}
p2++;
w2++;



}


while(*p3 == *w3){


	if(*p3 == '\0'){

	printf("Password found : %s\n", password3);
	return 1;

}
p3++;
w3++;



}


while(*p4 == *w4){


	if(*p4 == '\0'){

	printf("Password found : %s\n", password4);
	return 1;

}
p4++;
w4++;



}


return 0;



}


__global__ void kernel(){

char alphabet[26] = {'A', 'B', 'C', 'D', 'E', 'F', 'G', 'H', 'I', 'J', 'K', 'L', 'M', 'N', 'O', 'P', 'Q', 'S', 'T','U', 'V', 'W', 'X', 'Y', 'Z'};


char numbers[10] = {'0','1','2','3','4','5','6','7','8','9'};

char password[7];
password[6] = '\0';

int a, b, c, d;

for(a=0;a<10;a++){

	for(b=0;b<10;b++){

		for(c=0;c<10;c++){

			for(d=0;d<10;d++){

				password[0] = alphabet[blockIdx.x];
				password[1] = alphabet[threadIdx.x];
				password[2] = numbers[a];
				password[3] = numbers[b];
				password[4] = numbers[c];
				password[5] = numbers[d];

				if(is_a_match(password)){

					printf("Password found: %s\n", password);
}
}
			


}



}

}




}



int time_difference(struct timespec *start, struct timespec *finish, 
                              long long int *difference) {
  long long int ds =  finish->tv_sec - start->tv_sec; 
  long long int dn =  finish->tv_nsec - start->tv_nsec; 

  if(dn < 0 ) {
    ds--;
    dn += 1000000000; 
  } 
  *difference = ds * 1000000000 + dn;
  return !(*difference > 0);
}

int main(int argc, char *argv[]) {
  struct timespec start, finish;   
  long long int time_elapsed;

  clock_gettime(CLOCK_MONOTONIC, &start);
 
 kernel <<<26, 26>>>();
hipDeviceSynchronize();

  clock_gettime(CLOCK_MONOTONIC, &finish);
  time_difference(&start, &finish, &time_elapsed);
  printf("Time elapsed was %lldns or %0.9lfs\n", 
    time_elapsed, (time_elapsed/1.0e9)); 

  return 0;
}
